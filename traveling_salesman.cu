#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <limits.h>

#define CITIES 5 
#define N_POPULATION 512
#define GENERATIONS 10
#define MUTATION_RATE 0.05
#define TOURNAMENT_SIZE 5  

typedef struct {
    int path[CITIES];
    int fitness;
} Individual;

__managed__ Individual *population;
__managed__ Individual *new_population;
__managed__ hiprandState *devStates;

void generate_cities_distances(int *matrix) {
    for (int i = 0; i < CITIES; ++i) {
        for (int j = i + 1; j < CITIES; ++j) {
            int random_distance = rand() % 100 + 1;
            matrix[i * CITIES + j] = random_distance;
            matrix[j * CITIES + i] = random_distance;
        }
    }
}

void print_matrix(int *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d ", *(matrix + i * CITIES + j));
        }
        printf("\n");
    }
}

__global__ void initialize_population(Individual *population, hiprandState *state) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N_POPULATION) {
        hiprand_init(1234, idx, 0, &state[idx]);

        for (int i = 0; i < CITIES; i++) {
            population[idx].path[i] = i;
        }

        for (int i = CITIES; i > 1; --i) {
            int j = 1 + hiprand(&state[idx]) % i;
            int temp = population[idx].path[i];
            population[idx].path[i] = population[idx].path[j];
            population[idx].path[j] = temp;
        }

        population[idx].fitness = INT_MAX;
    }
}

__device__ bool is_valid_path(int *path) {
    bool visited[CITIES] = {false};  
    for (int i = 0; i < CITIES; ++i) {
        int city = path[i];
        if (city < 0 || city >= CITIES || visited[city]) {
            return false;  
        }
        visited[city] = true;
    }
    return true;
}

__device__ int calculate_permutation_cost(int *distances, int *path) {
    if (!is_valid_path(path)) {
        return INT_MAX;  
    }

    int cost = 0;
    for (int i = 0; i < CITIES - 1; ++i) {
        cost += distances[path[i] * CITIES + path[i + 1]];
    }
    cost += distances[path[CITIES - 1] * CITIES + path[0]];  
    return cost;
}

__global__ void evaluate_fitness(int *distances, Individual *population) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N_POPULATION) {
        population[idx].fitness = calculate_permutation_cost(distances, population[idx].path);
    }
}

__device__ void crossover(Individual *parent1, Individual *parent2, Individual *child) {
    int start = blockIdx.x % CITIES;
    int end = (start + CITIES / 2) % CITIES;

    for (int i = start; i != end; i = (i + 1) % CITIES) {
        child->path[i] = parent1->path[i];
    }

    int pos = end;
    for (int i = 0; i < CITIES; ++i) {
        bool found = false;
        for (int j = start; j != end; j = (j + 1) % CITIES) {
            if (parent2->path[i] == child->path[j]) {
                found = true;
                break;
            }
        }
        if (!found) {
            child->path[pos] = parent2->path[i];
            pos = (pos + 1) % CITIES;
        }
    }
}

__device__ void mutate(Individual *ind, hiprandState *states) {
    if (hiprand_uniform(&states[threadIdx.x]) < MUTATION_RATE) {
        int i = hiprand(&states[threadIdx.x]) % CITIES;
        int j = hiprand(&states[threadIdx.x]) % CITIES;
        int temp = ind->path[i];
        ind->path[i] = ind->path[j];
        ind->path[j] = temp;
    }
}

__device__ Individual* tournament_selection(Individual *population, hiprandState *states) {
    int best_idx = hiprand(&states[threadIdx.x]) % N_POPULATION;
    for (int i = 0; i < TOURNAMENT_SIZE; ++i) {
        int competitor_idx = hiprand(&states[threadIdx.x]) % N_POPULATION;
        if (population[competitor_idx].fitness < population[best_idx].fitness) {
            best_idx = competitor_idx;
        }
    }
    return &population[best_idx];
}

__global__ void generate_new_population(Individual *population, Individual *new_population, hiprandState *states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N_POPULATION) {
        Individual *parent1 = tournament_selection(population, states);
        Individual *parent2 = tournament_selection(population, states);

        crossover(parent1, parent2, &new_population[idx]);
        mutate(&new_population[idx], states);
    }
}

int main() {
    int *cities_distances;
    hipMallocManaged(&cities_distances, CITIES * CITIES * sizeof(int));
    hipMallocManaged(&population, N_POPULATION * sizeof(Individual));
    hipMallocManaged(&new_population, N_POPULATION * sizeof(Individual));
    hipMallocManaged(&devStates, N_POPULATION * sizeof(hiprandState));

    generate_cities_distances(cities_distances);
    print_matrix(cities_distances, CITIES, CITIES);

    int threadsPerBlock = 32;
    int blocks = (N_POPULATION + threadsPerBlock - 1) / threadsPerBlock;

    initialize_population<<<blocks, threadsPerBlock>>>(population, devStates);
    hipDeviceSynchronize();

    int best_fitness = INT_MAX;
    int best_idx = -1;

    for (int gen = 0; gen < GENERATIONS; ++gen) {
        evaluate_fitness<<<blocks, threadsPerBlock>>>(cities_distances, population);
        hipDeviceSynchronize();

        generate_new_population<<<blocks, threadsPerBlock>>>(population, new_population, devStates);
        hipDeviceSynchronize();
        
        for (int i = 0; i < N_POPULATION; ++i) {
            if (population[i].fitness < best_fitness) {
                best_fitness = population[i].fitness;
                best_idx = i;
            }
        }

        Individual *temp = population;
        population = new_population;
        new_population = temp;
    }


    printf("Melhor Rota: ");
    for (int i = 0; i < CITIES; ++i) {
        printf("%d ", population[best_idx].path[i]);
    }
    printf("\nCusto: %d\n", best_fitness);

    hipFree(cities_distances);
    hipFree(population);
    hipFree(new_population);
    hipFree(devStates);

    return 0;
}

